#include <thread>
#include <chrono>
#include <vector>
#include <algorithm>
#include <assert.h>
#include <iostream>
#include "stdio.h"
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *a, int *b, int *c, int N)
{
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (threadId < N)
    {
        c[threadId] = a[threadId] + b[threadId];
    }
}

void verify_results(std::vector<int> a, std::vector<int> b, std::vector<int> c)
{
    for (int i = 0; i < a.size(); i++)
    {
        assert(c[i] == a[i] + b[i]);
    }
}

int main()
{
    constexpr int N = 1 << 16;
    size_t bytes = sizeof(int) * N;

    std::vector<int> a(N);
    std::vector<int> b(N);
    std::vector<int> c(N);

    std::generate(std::begin(a), std::end(a), []()
                  { return rand() % 100; });
    std::generate(std::begin(b), std::end(b), []()
                  { return rand() % 100; });

    int *device_a, *device_b, *device_c;
    hipError_t err = hipSuccess;
    err = hipMalloc(&device_a, bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc(&device_b, bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc(&device_c, bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(device_a, a.data(), bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy vector A from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(device_b, b.data(), bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy vector B from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int NUM_THREADS = 1 << 8; // 1024

    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    printf("CUDA kernel launch with %d blocks of %d threads\n", NUM_BLOCKS,
           NUM_THREADS);
    vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(device_a, device_b, device_c, N);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(c.data(), device_c, bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy vector C from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    verify_results(a, b, c);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}