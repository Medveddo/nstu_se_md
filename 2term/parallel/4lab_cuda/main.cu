#include <chrono>
#include <vector>
#include <algorithm>
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <numeric>
// #include <thread> 

__global__ void countTargetLengthSequences(int *a, int targetLength, int N, int* totalSum)
{
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (threadId >= N) {return;} // boundary check

    int currentValue = a[threadId];
    int iterationsCount = 1;
    while (currentValue != 1 && currentValue > 0) // for integer overflow (e.g. 113383)
    {
        iterationsCount++;
        currentValue = currentValue % 2 == 0 ? currentValue / 2 : 3 * currentValue + 1;
    }
    if (currentValue < 0) {return;}
    if (iterationsCount == targetLength) {
        atomicAdd(totalSum, 1);
    }
}

int main()
{
    auto startTime = std::chrono::steady_clock::now();
    int intervalStart = 1;
    int intervalEnd = 1000000;
    int targetLength = 52;

    int foundSequences = 0;

    int N = (intervalEnd - intervalStart);
    size_t bytes = sizeof(int) * N;

    std::vector<int> a(N);

    for (int i = intervalStart; i < intervalEnd; i++) {
        a[i] = i;
    }
    int *device_a, *device_totalSum;
    hipError_t err = hipSuccess;
    
    err = hipMalloc(&device_a, bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc(&device_totalSum, sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device device_totalSum (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    auto startKernelTime = std::chrono::steady_clock::now();
    err = hipMemcpy(device_a, a.data(), bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy vector A from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(device_totalSum, &foundSequences, sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy totalSum from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int NUM_THREADS = 1024;

    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    printf("CUDA kernel launch with %d blocks of %d threads\n", NUM_BLOCKS,
           NUM_THREADS);

    countTargetLengthSequences<<<NUM_BLOCKS, NUM_THREADS>>>(device_a, targetLength, N, device_totalSum);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(&foundSequences, device_totalSum, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to memcpy from device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    auto endKernelTime = std::chrono::steady_clock::now();

    hipFree(device_a);
    hipFree(device_totalSum);

    auto endTime = std::chrono::steady_clock::now();
    const auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime -
                                                            startTime).count();
    const auto durationKernel = std::chrono::duration_cast<std::chrono::microseconds>(endKernelTime -
                                                            startKernelTime).count();
    std::cout << "(CUDA_KernelOnly) - Interval[" << intervalStart << ":" << intervalEnd << "]Len["<<targetLength<<"] Found: "
    << foundSequences << " sequences Took: " << durationKernel << "[µs]"<< std::endl;
    std::cout << "(CUDA_Total) - Interval[" << intervalStart << ":" << intervalEnd << "]Len["<<targetLength<<"] Found: "
    << foundSequences << " sequences Took: " << duration << "[µs]"<< std::endl;
    return 0;
}