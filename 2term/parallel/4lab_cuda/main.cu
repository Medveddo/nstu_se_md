#include <chrono>
#include <vector>
#include <algorithm>
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <numeric>

__global__ void countTargetLengthSequences(int *a, int *c, int targetLength, int N)
{
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (threadId >= N) {return;} // boundary check

    int currentValue = a[threadId];
    int iterationsCount = 1;
    while (currentValue != 1 && currentValue > 0) // for integer overflow (e.g. 113383)
    {
        iterationsCount++;
        currentValue = currentValue % 2 == 0 ? currentValue / 2 : 3 * currentValue + 1;
    }
    if (currentValue < 0) {return;}
    if (iterationsCount == targetLength) {
        c[threadId] = 1;
    }
}

int main()
{
    auto startTime = std::chrono::steady_clock::now();
    int intervalStart = 1;
    int intervalEnd = 250000;
    int targetLength = 24;

    int foundSequences = 0;

    int N = (intervalEnd - intervalStart);
    size_t bytes = sizeof(int) * N;

    std::vector<int> a(N);
    std::vector<int> c(N);

    for (int i = intervalStart; i < intervalEnd; i++) {
        a[i] = i;
    }
    int *device_a, *device_c;
    hipError_t err = hipSuccess;
    err = hipMalloc(&device_a, bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc(&device_c, bytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    auto startKernelTime = std::chrono::steady_clock::now();
    err = hipMemcpy(device_a, a.data(), bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy vector A from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int NUM_THREADS = 1024;

    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    printf("CUDA kernel launch with %d blocks of %d threads\n", NUM_BLOCKS,
           NUM_THREADS);


    countTargetLengthSequences<<<NUM_BLOCKS, NUM_THREADS>>>(device_a, device_c, targetLength, N);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(c.data(), device_c, bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy vector C from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    auto endKernelTime = std::chrono::steady_clock::now();
    
    foundSequences = std::accumulate(c.begin(), c.end(), 0);
    auto endKernelAndAccumTime = std::chrono::steady_clock::now();
    
    hipFree(device_a);
    hipFree(device_c);

    auto endTime = std::chrono::steady_clock::now();
    const auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime -
                                                            startTime).count();
    const auto durationKernel = std::chrono::duration_cast<std::chrono::microseconds>(endKernelTime -
                                                            startKernelTime).count();
    const auto durationKernelAccum = std::chrono::duration_cast<std::chrono::microseconds>(endKernelAndAccumTime -
                                                            startKernelTime).count();
    std::cout << "(CUDA_KernelOnly) - Interval[" << intervalStart << ":" << intervalEnd << "]Len["<<targetLength<<"] Found: "
    << foundSequences << " sequences Took: " << durationKernel << "[µs]"<< std::endl;
    std::cout << "(CUDA_KernelAndAccum) - Interval[" << intervalStart << ":" << intervalEnd << "]Len["<<targetLength<<"] Found: "
    << foundSequences << " sequences Took: " << durationKernelAccum << "[µs]"<< std::endl;
    std::cout << "(CUDA_Total) - Interval[" << intervalStart << ":" << intervalEnd << "]Len["<<targetLength<<"] Found: "
    << foundSequences << " sequences Took: " << duration << "[µs]"<< std::endl;
    return 0;
}